#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addOne(int* d_array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_array[idx] += 1;
    }
}

int main() {
    const int arraySize = 10;
    const int arrayBytes = arraySize * sizeof(int);

    // Allocate memory on the host (CPU)
    int h_array[arraySize] = {0};

    // Allocate memory on the device (GPU)
    int* d_array;
    hipMalloc((void**)&d_array, arrayBytes);

    // Transfer the array from the host to the device
    hipMemcpy(d_array, h_array, arrayBytes, hipMemcpyHostToDevice);

    // Launch the kernel with 1 block of 10 threads
    addOne<<<1, arraySize>>>(d_array, arraySize);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Transfer the array from the device back to the host
    hipMemcpy(h_array, d_array, arrayBytes, hipMemcpyDeviceToHost);

    // Print the resulting array on the host
    printf("Resulting array:\n");
    for (int i = 0; i < arraySize; i++) {
        printf("%d ", h_array[i]);
    }
    printf("\n");

    // Free the memory allocated on the device
    hipFree(d_array);

    return 0;
}

