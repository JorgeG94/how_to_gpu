#include "hip/hip_runtime.h"
// multi_gpu_hello_world.cu
// A simple CUDA program to demonstrate multi-GPU "Hello, World!" using MPI.

#include <mpi.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to print "Hello, World!" from each GPU
__global__ void helloWorldFromGPU(int gpu_id, int rank) {
    printf("Hello, World! from GPU %d, process rank %d, thread %d\n", gpu_id, rank, threadIdx.x);
}

int main(int argc, char** argv) {
    // Initialize the MPI environment
    MPI_Init(&argc, &argv);

    // Get the number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Get the rank of the process
    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    // Get the number of GPUs
    int num_gpus;
    hipGetDeviceCount(&num_gpus);

    // Assign a GPU to each MPI process
    int gpu_id = world_rank % num_gpus;
    hipSetDevice(gpu_id);

    // Print information from each GPU
    helloWorldFromGPU<<<1, 10>>>(gpu_id, world_rank);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Finalize the MPI environment
    MPI_Finalize();

    return 0;
}

