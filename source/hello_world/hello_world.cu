#include <stdio.h>
#include <hip/hip_runtime.h>
// Kernel function to print "Hello, World!" from the GPU
__global__ void helloWorldFromGPU() {
  printf("Hello, World! from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {

  // Launch the kernel with 2 block of 5 threads
  helloWorldFromGPU<<<2, 5>>>();
  
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
    
  return 0;
}

